/* Monte Carlo simulation of the Ising model using CUDA */
/* Author: Jorge Fernandez de Cossio Diaz */
/* March, 2019 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
//#include <curand.h>

#define RANDSEED 5  // random seed

/* Linear dimension of square grid. The block-size in my laptop is 1024. 
Therefore we set L=32 so that there are a total of L^2 = 1024 spins. */
#define L 32	// linear dimension of square grid.
#define N (L*L)	// total number of spins

#define ITERATIONS 10001	// number of iterations

/* linear index of current block */
__device__ int globalBlockIdx() {
    return blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
}

/* linear index of current thread inside its block */
__device__ int threadIdxInBlock() {
	return threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
}

/* total number of threads per block */
__device__ int blockVolume() {
    return blockDim.x * blockDim.y * blockDim.z;
}

/* gobal linear index of current thread */
__device__ int globalThreadIdx() {
    return threadIdxInBlock() + globalBlockIdx() * blockVolume();
}

/* setups random number generation in each thread */
__global__ void initialize_rand(hiprandState *rngState) {
	// Each thread gets same seed, a different sequence number, no offset
	int idx = globalThreadIdx();
	assert(0 <= idx && idx < N);
	hiprand_init(RANDSEED, idx, 0, rngState + idx);
}

/* random uniform real in [0,1] */
__device__ float randreal(hiprandState *rngState) {
	return hiprand_uniform(rngState + globalThreadIdx());
}

/* returns random -1 or +1 */
__device__ short randspin(hiprandState *rngState) {
	return 2 * (short)roundf(randreal(rngState)) - 1;
}

/* returns linear index corresponding to Cartesian index x, y 
in the periodic square grid */
__host__ __device__ short linear_index(int x, int y) {
	while (x < 0)  {x += L; }
	while (x >= L) {x -= L; }
	while (y < 0)  {y += L; }
	while (y >= L) {y -= L; }
	assert(x >= 0 && x < L);
	assert(y >= 0 && y < L);
	int idx = x + L * y;
	assert(0 <= idx && idx < N);
	return idx;
}

__host__ __device__ short get(short *spins, int x, int y) {
	return spins[linear_index(x, y)];
}

__host__ __device__ short set(short *spins, int x, int y, short state) {
	spins[linear_index(x, y)] = state;
	return state;
}

/* initializes the spins to random states */
__global__ void initialize_spins(short *spins, hiprandState *rngState) {
	spins[globalThreadIdx()] = randspin(rngState);
}

/* sum of neighboring spins */
__device__ short neighbor_sum(short *spins) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	short sum = 0;
	sum += get(spins, x - 1, y);
	sum += get(spins, x + 1, y);
	sum += get(spins, x, y - 1);
	sum += get(spins, x, y + 1);
	return sum;
}

/* update all spins with metropolis rule at inverse temperature beta. */
__device__ void metropolis(short *spins, hiprandState *rngState, float beta) {
	int idx = globalThreadIdx();
	short state = spins[idx];
	assert(state == 1 || state == -1);
	
	// Metropolis update rule
	float deltaE = 2.0f * state * neighbor_sum(spins);
	//if (idx == 10) { printf("thread %i, beta %f, deltaE %f, expf(-beta * deltaE) %f\n", idx, beta, deltaE, expf(-beta * deltaE)); }
	//if (deltaE <= 0) { assert(randreal(rngState) < expf(-beta * deltaE)); }
	float u = randreal(rngState);
	if (u < 0.5 && u < expf(-beta * deltaE)) {
		state = -state;
	}

	// synchronous update
	__syncthreads();	// wait for all threads to compute new state
	spins[idx] = state;
	__syncthreads();	// wait for all threads to update spins
}

/* copies spin states between two arrays in device memory */
__device__ void spinsCpy(short *to, short *from) {
	int idx = globalThreadIdx();
	to[idx] = from[idx];
	__syncthreads();
}

/* simulates the system of spins in shared memory */
__global__ void simulate(short *spinsGlob, hiprandState *rngState, float beta) {
	__shared__ short spinsShared[N];
	// copy spins from global to shared memory
	spinsCpy(spinsShared, spinsGlob);
	
	// simulate
	for (int iter = 0; iter < ITERATIONS; ++iter) {
		metropolis(spinsShared, rngState, beta);
	}

	// copy spins back to global memory
	spinsCpy(spinsGlob, spinsShared);
}

/* return magnetization of system of spins */
__host__ __device__ float magnetization(short* spins) {
	short M = 0;
	for (int i = 0; i < N; ++i) {
		assert(spins[i] == -1 || spins[i] == 1);
		M += spins[i];
	}
	return (float)M / N;
}

/* prints the grid of spins */
__host__ __device__ void print_spins(short *spins) {
	for (int x = 0; x < L; ++x) {
		for (int y = 0; y < L; ++y) {
			short s = get(spins, x, y);
			if (s == 1) {
				printf("+ ");
			} else if (s == -1) {
				printf("- ");
			} else {
				printf("%i ", s);
			}
		}
		printf("\n");
	}
}

int main(void) {
	printf("Simulating %i spins, on a square grid of length %i\n", N, L);

	// random setup
	hiprandState *rngStatesDev;
	hipMalloc(&rngStatesDev, N * sizeof(hiprandState));

	dim3 blockSize(L,L);
	initialize_rand<<<1, blockSize>>>(rngStatesDev);

	// allocate host/device memory for spins
	short *spins;
	hipMallocManaged(&spins, N * sizeof(short));

	// initialize spins to random configurations
	initialize_spins<<<1, blockSize>>>(spins, rngStatesDev);
	hipDeviceSynchronize();

	//printf("Initialized spins (in random state), |m| = %f\n", abs(magnetization(spins)));
	//print_spins(spins);
	
	// simulate
	printf("beta\tabsolute magnetization\n");
	for (float beta = 0.0f; beta <= 1.0f; beta += 0.01f) {
		simulate<<<1, blockSize>>>(spins, rngStatesDev, beta);
		hipDeviceSynchronize();

		float m = magnetization(spins);
		printf("%f\t%f\n", beta, abs(m));
		fflush(stdout);
	}

	//printf("Final configuration, |m| = %f\n", abs(magnetization(spins)));
	//print_spins(spins);

	hipFree(rngStatesDev);
	hipFree(spins);

	return 0;
}